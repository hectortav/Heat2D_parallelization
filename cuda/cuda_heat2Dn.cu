#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define NXPROB      20480                /* x dimension of problem grid */
#define NYPROB      32768                  /* y dimension of problem grid */
#define STEPS       500                /* number of time steps */
#define MAXWORKER   8                  /* maximum number of worker tasks */
#define MINWORKER   3                  /* minimum number of worker tasks */
#define BEGIN       1                  /* message tag */
#define LTAG        2                  /* message tag */
#define RTAG        3                  /* message tag */
#define NONE        0                  /* indicates no neighbor */
#define DONE        4                  /* message tag */
#define MASTER      0                  /* taskid of first process */

#define BLOCK_H   10
#define BLOCK_V   8
#define THREADS   32

struct Parms {
  float cx;
  float cy;
} parms = {0.1, 0.1};

/**************************************************************************
 *  subroutine update
 ****************************************************************************/
void update(int start, int end, int ny, float *u1, float *u2)
{
   int ix, iy;
   for (ix = start; ix <= end; ix++)
      for (iy = 1; iy <= ny-2; iy++)
         *(u2+ix*ny+iy) = *(u1+ix*ny+iy)  +
                          parms.cx * (*(u1+(ix+1)*ny+iy) +
                          *(u1+(ix-1)*ny+iy) -
                          2.0 * *(u1+ix*ny+iy)) +
                          parms.cy * (*(u1+ix*ny+iy+1) +
                         *(u1+ix*ny+iy-1) -
                          2.0 * *(u1+ix*ny+iy));
}

/*****************************************************************************
 *  subroutine inidat
 *****************************************************************************/
void inidat(int nx, int ny, float *u) {
int ix, iy;
for (ix = 0; ix <= nx-1; ix++)
  for (iy = 0; iy <= ny-1; iy++)
     {*(u+ix*ny+iy) = (float)(ix * (nx - ix - 1) * iy * (ny - iy - 1)%1000);
     //if (*(u+ix*ny+iy) > 10000.0)
     //printf("%f\n", *(u+ix*ny+iy));
    }
}

/**************************************************************************
 * subroutine prtdat
 **************************************************************************/
void prtdat(int nx, int ny, float *u1, const char *fnam) {
int ix, iy;
FILE *fp;

fp = fopen(fnam, "w");
for (iy = ny-1; iy >= 0; iy--) {
  for (ix = 0; ix <= nx-1; ix++) {
    fprintf(fp, "%6.1f", *(u1+ix*ny+iy));
    if (ix != nx-1)
      fprintf(fp, " ");
    else
      fprintf(fp, "\n");
    }
  }
fclose(fp);
}

__global__ void cuda_update(float *u0, float *u1, struct Parms parms)
{
  int ix, iy;
  ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
  iy = blockIdx.y * blockDim.y + threadIdx.y + 1;

  if (ix > 0 && iy > 0)
  {
    if (ix + iy < NXPROB + NYPROB - 2)
    {
      *(u1+ix*NYPROB+iy) = *(u0+ix*NYPROB+iy)  +
                          parms.cx * (*(u0+(ix+1)*NYPROB+iy) +
                          *(u0+(ix-1)*NYPROB+iy) -
                          2.0 * *(u0+ix*NYPROB+iy)) +
                          parms.cy * (*(u0+ix*NYPROB+iy+1) +
                         *(u0+ix*NYPROB+iy-1) -
                          2.0 * *(u0+ix*NYPROB+iy));
    }
  }
}

__global__ void MyKernel(int *a, int *b, int *c, int N) 
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < N) 
  {
    c[idx] = a[idx] + b[idx];
  } 
} 

int main (int argc, char *argv[])
{
  int i;
  float *u;
  float *cuda_u0, *cuda_u1;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float ms = 0.0f;
  int block_size;
  int min_grid, grid;
  
  //http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
  hipOccupancyMaxPotentialBlockSize(&min_grid, &block_size, MyKernel, 0, NXPROB*NYPROB);
  grid = (NXPROB*NYPROB + block_size - 1) / block_size;
  dim3 dimBlocks(BLOCK_H, BLOCK_V);
  dim3 dimThreads((NXPROB / BLOCK_H) + ((NXPROB % BLOCK_H) != 0), (NYPROB / BLOCK_V) + ((NYPROB % BLOCK_V) != 0));

  //malloc host 
  u = (float*)malloc(NXPROB*NYPROB*sizeof(float));

  //malloc device
  hipMalloc((void**)&cuda_u0, (NXPROB*NYPROB*sizeof(float)));
  hipMalloc((void**)&cuda_u1, (NXPROB*NYPROB*sizeof(float)));

  printf("Grid size: X= %d  Y= %d  Time steps= %d\n",NXPROB,NYPROB,STEPS);
  inidat(NXPROB, NYPROB, u); //initialize
  //prtdat(NXPROB, NYPROB, u, "initial.dat"); //print

  //copy from host to device
  hipMemcpy(cuda_u0, u, (NXPROB*NYPROB*sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(cuda_u1, u, (NXPROB*NYPROB*sizeof(float)), hipMemcpyHostToDevice);

  hipEventRecord(start, 0);
  for (i = 0; i < STEPS; i+=2)
  { 
      cuda_update<<<grid, block_size>>>(cuda_u0, cuda_u1, parms);
      cuda_update<<<grid, block_size>>>(cuda_u1, cuda_u0, parms);
  }
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);

  //copy from device to host
  hipMemcpy(u, cuda_u1, (NXPROB*NYPROB*sizeof(float)), hipMemcpyDeviceToHost);
  //prtdat(NXPROB, NYPROB, u, "final.dat");   //print
  printf("Time: %f ms\n", ms);
  
  hipFree(cuda_u0);
  hipFree(cuda_u1);
  free(u);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
